#include "hip/hip_runtime.h"
# matlab CUDA C codes for knn calculation. Author： Heng Zhang ######

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define N 500
#define DIM 21

__device__ float gpuCalDis(float *A, float *B)
{
	float dis = 0.0;
	float temp;
    #pragma unroll
	for (int i = 0; i < DIM; i++)
	{
		temp = A[i] - B[i];
		dis += temp*temp;
	}
	return dis;
}

__device__ int gpuCmpQueue(float *queue_dis, float dis)
{
	if (queue_dis[N - 1] <= dis)
		return -1;                                     //Do not need to update the queue
	else
	{
		int index = 0;
		while (queue_dis[index] <= dis)
		{
			index++;
		}
		return index;                                  //Need to update the queue, return the position
	}
}

__device__ void gpuUpdateQueue(int *queue_id, float *queue_dis, int id, float dis)
{
	int index = gpuCmpQueue(queue_dis, dis);
	if (index != -1)
	{
		int i = N - 1;
		while (i > index)
		{
			queue_id[i] = queue_id[i - 1];
			queue_dis[i] = queue_dis[i - 1];
			i--;
		}
		queue_id[index] = id;
		queue_dis[index] = dis;
	}
}

__device__ void gpuCopyRecord(float *Array, float *X, int index)
{
	int offset = index*DIM;
    #pragma unroll
	for (int i = 0; i < DIM; i++)
	{
		X[i] = Array[offset + i];
	}
}

__device__ void gpuFillResult(int *Result, int *queue_id, int index)
{
	int offset = index*N;
    #pragma unroll
	for (int i = 0; i < N; i++)
	{
		Result[offset + i] = queue_id[i];
	}
}

__device__ void gpuFillDis(float *Dis, float *queue_dis, int index)
{
	int offset = index*N;
	for (int i = 0; i < N; i++)
	{
		Dis[offset + i] = queue_dis[i];
	}
}

__global__ void gpuFindNearestPoints(int *Result,  float *Array, int total, int start, int stop)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x + start;
	if (i >= stop)
		return;
	else
	{
		float A[DIM] = { 0.0 };
		float B[DIM] = { 0.0 };
		float dis = 0.0;
		int queue_id[N] = { 0 };
		float queue_dis[N];
        #pragma unroll
		for (int k = 0; k < N; k++)
			queue_dis[k] = 9999.0;
		gpuCopyRecord(Array, A, i);
		for (int j = 0; j < total; j++)
		{
			if (j == i)
				continue;
			else
			{
				gpuCopyRecord(Array, B, j);
				dis = gpuCalDis(A, B);
				gpuUpdateQueue(queue_id, queue_dis, j, dis);
			}
		}
		gpuFillResult(Result, queue_id, i-start);
      
	}
}
